#include "hip/hip_runtime.h"
#include <stdio.h>

#include <EdbDataSet.h>
#include <TGraph.h>
#include <TH2.h>
#include <TNtupleD.h>
#include <TObjArray.h>
#include <EdbPattern.h>
#include <TFitter.h>
#include <TVirtualFitter.h>
#include <TMath.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

const int NPIDMAX=800;
TObjArray *gTracks;
TVirtualFitter *gMinuit;
int gpidMax;
int gpidMin;
int gpidcutntrk;
int gflag_chi2dis;
float robustfactor;
double p[NPIDMAX*2] ={};
int nPID;
int ncall = 0;
const double XYrange = 8500;

struct cudaSegment{
	int flag, pid;
	float x,y,z;
};


struct cudaTrack{
	float x,y,z,tx,ty,tx_first8,ty_first8;
	int nseg;
	cudaSegment segments[NPIDMAX];
};

// Data buffer for the GPU process
cudaTrack* h_tracks;
double *h_params;
float* h_chi2;
cudaTrack* d_tracks;
float* d_chi2;
double *d_params;
int* indexArray;





// 最小二乗法の計算 
__global__ void lsm_kernel(int n, cudaTrack* d_trk, double *d_param) {
	// access thread id
	const unsigned int tid = threadIdx.x;
	const unsigned int tsize = blockDim.x;
	const unsigned int bid = blockIdx.x;
	// y = a0 + a1*x
	// x = a0 + a1*z
	int pos = tid + tsize*bid;
	if (pos < n) {
		cudaTrack *t = &d_trk[pos];
        int i;
        double A00=0 ,A01=0, A02=0, A11=0, A12=0;
        double B00=0 ,B01=0, B02=0, B11=0, B12=0;
        for (i=0;i<NPIDMAX;i++) {
			cudaSegment *s = &t->segments[i];
	 		float x = s->x + d_param[s->pid*2];
	 		float y = s->y + d_param[s->pid*2+1];
	 		float z = s->z;
			if(s->flag){
                A00+=1.0;
                A01+=z;
                A02+=x;
                A11+=z*z;
                A12+=z*x;
                B00+=1.0;
                B01+=z;
                B02+=y;
                B11+=z*z;
                B12+=z*y;
            }
        }
 
        t->x = (A02*A11-A01*A12) / (A00*A11-A01*A01);
        t->tx = (A00*A12-A01*A02) / (A00*A11-A01*A01);

        t->y = (B02*B11-B01*B12) / (B00*B11-B01*B01);
        t->ty = (B00*B12-B01*B02) / (B00*B11-B01*B01);
        t->z = 0;
	}
	__syncthreads();
}

void lsm(double x[],double y[], int N, double &a0, double &a1)
{
	// y = a0 + a1*x
    int i;
    double A00=0 ,A01=0, A02=0, A11=0, A12=0;

    for (i=0;i<N;i++) {
        A00+=1.0;
        A01+=x[i];
        A02+=y[i];
        A11+=x[i]*x[i];
        A12+=x[i]*y[i];
    }
    a0 = (A02*A11-A01*A12) / (A00*A11-A01*A01);
    a1 = (A00*A12-A01*A02) / (A00*A11-A01*A01);
}

	

__global__ void calc_chi2_kernel(int n, cudaTrack* d_trk, double *p, float *d_chi2) {
	// access thread id
	const unsigned int tid = threadIdx.x;
	const unsigned int tsize = blockDim.x;
	const unsigned int bid = blockIdx.x;
	
	int pos = tid + tsize*bid;
	float chi2 = 0;
	if (pos < n) {
		float sigmaPos2 = 0.36;//0.6 * 0.6;
		float sigmaAng2 = 4e-6;//0.002 * 0.002;
		
		cudaTrack *t = &d_trk[pos];
		
		int nseg=0;
		for(int i=0; i<NPIDMAX; i++)
		{
			cudaSegment *s = &t->segments[i];
			if(s->flag==0) continue;
			float x = s->x + p[i*2];
			float y = s->y + p[i*2+1];
			float z = s->z;
			float dx = x - (t->x+t->tx*(z-t->z));
			float dy = y - (t->y+t->ty*(z-t->z));
			chi2 += dx * dx + dy * dy;
			nseg++;
		}
		chi2 /= sigmaPos2*nseg;
		float txFit = t->tx;
		float tyFit = t->ty;
		float tx = t->tx_first8;
		float ty = t->ty_first8;
		chi2 += ((tx - txFit) * (tx - txFit) + (ty - tyFit) * (ty - tyFit)) / sigmaAng2;
		d_chi2[pos] = chi2;
		
	}
	__syncthreads();
}

void fitfuncRobust(Int_t &npar, Double_t *grad, Double_t &fval, Double_t *p, Int_t iflag)
{
	
	double delta2 = 0.0;
	
	for(int i=0; i<NPIDMAX*2; i++){h_params[i]=p[i];}
	
	checkCudaErrors( hipMemcpy( d_params, h_params, sizeof(double)*NPIDMAX*2, hipMemcpyHostToDevice) );
	
	int ntrk = gTracks->GetEntriesFast();
	int numthread = 512;
	int numblock = (ntrk + numthread -1)/numthread;
	dim3 threads(numthread, 1, 1);
	dim3 blocks(numblock, 1, 1);
	
	lsm_kernel <<< blocks, threads >>> (ntrk, d_tracks, d_params);
	hipDeviceSynchronize();
	// check if kernel execution generated and error
	getLastCudaError("lsm Kernel execution failed");
	
	/*if(ncall==0){
		checkCudaErrors( hipMemcpy( h_tracks, d_tracks, sizeof(cudaTrack)*ntrk, hipMemcpyDeviceToHost) );
		printf("h_track %f %f %f %f %f\n", h_tracks[0].x, h_tracks[0].y, h_tracks[0].z, h_tracks[0].tx,  h_tracks[0].ty);
	}*/
	
	calc_chi2_kernel<<< blocks, threads >>> (ntrk, d_tracks, d_params, d_chi2);
	hipDeviceSynchronize();
	getLastCudaError("calc chi2 Kernel execution failed");
	
	// thrust, sort on GPU
	thrust::sort(thrust::device, d_chi2, d_chi2+ntrk);
	checkCudaErrors( hipMemcpy( h_chi2, d_chi2, sizeof(float)*ntrk, hipMemcpyDeviceToHost) );
	// float robustfactor = 0.5; // only % of segments will be used.
	int nrobust = ntrk*robustfactor;
	for (int i = 0; i < nrobust; i++) 
	{
		delta2 += h_chi2[i];
	}
	double lambda = 0.1;
	
	//regularization
	/*for(int i=0;i<nPID*2)
	{
		delta2+=lambda*p[i]*p[i]; //L2 regularization
		// delta2+=lambda*abs(p[i]); //L1 regularization
	}*/
	fval = delta2;
	if(ncall%1000==0) {
		printf("ncall=%d fval = %lf ", ncall, fval);
		for(int i=0; i<10; i++){printf("%4.1lf ", p[i]);}
		printf("\n");
	}
	ncall++;
}


void calc_align_par(TObjArray *tracks,double iX, double iY, double bin_width, int fixflag, TNtupleD *sta)
{
	gTracks = tracks;
	int ntrk = gTracks->GetEntriesFast();
	// Cuda data buffers
	checkCudaErrors( hipHostMalloc( (void**) &h_tracks, sizeof(cudaTrack)*ntrk) );
	checkCudaErrors( hipHostMalloc( (void**) &h_chi2, sizeof(float)*ntrk) );
	checkCudaErrors( hipHostMalloc( (void**) &h_params, sizeof(double)*NPIDMAX*2) );
	
	checkCudaErrors( hipMalloc( (void**) &d_tracks, sizeof(cudaTrack)*ntrk) );
	checkCudaErrors( hipMalloc( (void**) &d_chi2, sizeof(float)*ntrk) );
	checkCudaErrors( hipMalloc( (void**) &d_params, sizeof(double)*NPIDMAX*2) );
	
	for (int i = 0; i < ntrk; i++)
	{
		EdbTrackP *t = (EdbTrackP *)gTracks->At(i);
		cudaTrack *ct = &h_tracks[i];
		ct->tx_first8 = t->TX();
		ct->ty_first8 = t->TY();
		ct->nseg = t->N();
		for(int ipid=0; ipid<NPIDMAX; ipid++){ ct->segments[ipid].flag=0;} //初期値クリア
		for (int iseg = 0; iseg < t->N(); iseg++)
		{
			EdbSegP *s = t->GetSegment(iseg);
			// if (1)
			if (fabs(s->X() - iX) < bin_width / 2 && fabs(s->Y() - iY) < bin_width / 2)
			{
				int pid = s->PID();
				ct->segments[pid].flag=1;
				ct->segments[pid].x=s->X();
				ct->segments[pid].y=s->Y();
				ct->segments[pid].z=s->Z();
			}
		}
	}
	checkCudaErrors( hipMemcpy( d_tracks, h_tracks, sizeof(cudaTrack)*ntrk, hipMemcpyHostToDevice) );
	// The default minimizer is Minuit, you can also try Minuit2
	TVirtualFitter::SetDefaultFitter("Minuit");
	// gMinuit->BuildArrays(30);
	// Int_t SetParameter(Int_t ipar, const char* parname, Double_t value, Double_t verr, Double_t vlow, Double_t vhigh)

	int pid=0; //最初のプレート
	gMinuit->SetParameter(2*pid,Form("dx%d",pid),0,0,0,0);
	gMinuit->SetParameter(2*pid+1,Form("dy%d",pid),0,0,0,0);
	for(pid=1;pid<nPID-1;pid++)
	{
		gMinuit->SetParameter(2*pid,Form("dx%d",pid),0,0.1,-30,30);
		gMinuit->SetParameter(2*pid+1,Form("dy%d",pid),0,0.1,-30,30);
	}
	pid=nPID-1; // 最後のプレート
	if(fixflag==1)
	{
		gMinuit->SetParameter(2*pid,Form("dx%d",pid),0,0,0,0);
		gMinuit->SetParameter(2*pid+1,Form("dy%d",pid),0,0,0,0);
	}else{
		gMinuit->SetParameter(2*pid,Form("dx%d",pid),0,0.1,-30,30);
		gMinuit->SetParameter(2*pid+1,Form("dy%d",pid),0,0.1,-30,30);
	}
	
	
	gMinuit->SetFCN(fitfuncRobust);
	
	
	double arglist[200];

	arglist[0] = 0;
	// set print level. arglist[0]==0 is minimum print.
	gMinuit->ExecuteCommand("SET PRIntout",arglist,1);

	// minimize
	arglist[0] = 200000; // number of function calls
	arglist[1] = 0.001; // tolerance
	gpidcutntrk = 0;
	gflag_chi2dis = 0;
	gMinuit->SetMaxIterations( 10000 );
	ncall =0;
	printf("Aligning iX = %.0f, iY = %.0f, ntrk = %d, robustfactor = %.1f\n",iX,iY,ntrk,robustfactor);
	gMinuit->ExecuteCommand("MIGRAD2", arglist, 2);
	/*
	gMinuit->SetFCN(fitfuncRobust3);
	gMinuit->ExecuteCommand("MIGRAD",arglist,2);
	*/
	
	/*
	double p[3];
	double parErrors[3];
	*/
	// get result
	for (int i = 0; i < nPID*2; ++i)
	{
		p[i] = gMinuit->GetParameter(i);

		// parErrors[i] = minuit->GetParError(i);
	}
	
	for(int pid=0;pid<nPID;pid++)
	{
		sta->Fill(iX,iY,p[pid*2],p[pid*2+1],pid);
	}
	
	checkCudaErrors( hipHostFree( h_tracks) );
	checkCudaErrors( hipHostFree( h_chi2) );
	checkCudaErrors( hipHostFree( h_params) );

	checkCudaErrors( hipFree( d_tracks) );
	checkCudaErrors( hipFree( d_chi2) );
	checkCudaErrors( hipFree( d_params) );
	

}

bool pass(EdbTrackP *t, double iX, double iY,double bin_width)
{
	for (int iseg = 0; iseg < t->N(); iseg++)
	{
		EdbSegP *s = t->GetSegment(iseg);
		if (fabs(s->X() - iX) < bin_width / 2 && fabs(s->Y() - iY) < bin_width / 2)
		{
			return true;
		}
	}
	return false;
}
int count_passed_seg(EdbTrackP *t, double iX, double iY, double bin_width)
{
	int count = 0;
	for (int iseg = 0; iseg < t->N(); iseg++)
	{
		EdbSegP *s = t->GetSegment(iseg);
		if (fabs(s->X() - iX) < bin_width / 2 && fabs(s->Y() - iY) < bin_width / 2)
			count++;
	}
	return count;
}
void apply_align(EdbTrackP *t, double iX, double iY,double bin_width)
{
	for (int iseg = 0; iseg < t->N(); iseg++)
	{
		EdbSegP *s = t->GetSegment(iseg);
		// if (1)
		if (fabs(s->X() - iX) < bin_width / 2 && fabs(s->Y() - iY) < bin_width / 2)
		{
			int pid = s->PID();
			s->SetX(s->X() + p[pid * 2]);
			s->SetY(s->Y() + p[pid * 2 + 1]);
		}
	}
}
int calc_dxy(EdbPVRec *pvr,TTree *tree, int ntrk, double Xcenter, double Ycenter, double bin_width)
{
	
	double deltaX, deltaY, tx, ty, deltaTX, deltaTY, x_t, y_t, slopeX, slopeY;
	int plate, cross_the_line,trid,nseg;
	tree->Branch("deltaX", &deltaX);
	tree->Branch("deltaY", &deltaY);
	tree->Branch("deltaTX", &deltaTX);
	tree->Branch("deltaTY", &deltaTY);
	tree->Branch("x", &x_t);
	tree->Branch("y", &y_t);
	tree->Branch("slopeX", &slopeX);
	tree->Branch("slopeY", &slopeY);
	tree->Branch("pl", &plate);
	tree->Branch("cross_the_line", &cross_the_line);
	tree->Branch("trid", &trid);
	tree->Branch("nseg", &nseg);
	double tx3;
	double stx;
	double ty3;
	double sty;
	// Loop over the tracks
	for (int itrk = 0; itrk < ntrk; itrk++)
	{

		stx = 0.;
		sty = 0.;
		EdbTrackP *t = pvr->GetTrack(itrk);
		if (abs(t->TX() + 0.01) >= 0.01 || abs(t->TY() - 0.004) >= 0.01 || t->N() < 5)
			continue;

		trid=t->ID();
		nseg = t->N();
		TGraph grX;
		TGraph grY;
		int pl;
		int plold = 10000;
		int consecutive_seg = 0;

		std::vector<double> vx(nseg);
		std::vector<double> vy(nseg);
		std::vector<double> vz(nseg);
		std::vector<double> vtx(nseg);
		std::vector<double> vty(nseg);

		double x3, y3, z3;
		for(int iPID=2;iPID<nPID-2;iPID++)
		{
			int count=0;
			double x[5];
			double y[5];
			double z[5];
			for (int iseg = 0; iseg < nseg; iseg++)
			{
				EdbSegP *s = t->GetSegment(iseg);
				for(int ipoint=0;ipoint<5;ipoint++)
				{
					if(s->PID()==iPID-2+ipoint)
					{
						x[ipoint]=s->X();
						y[ipoint]=s->Y();
						z[ipoint]=s->Z();
						count++;
					}
				}
				if(s->PID()==iPID)
				{
					tx3=s->TX();
					ty3=s->TY();
				}
			}
			if(count!=5) continue;
			// printf("%d\n",grX.GetN());
			int areaX[5];
			int areaY[5];
			for (int ipoint = 0; ipoint < 5; ipoint++)
			{
				areaX[ipoint] = (x[ipoint] - (Xcenter - XYrange)) / bin_width;
				areaY[ipoint] = (y[ipoint] - (Ycenter - XYrange)) / bin_width;
			}
			cross_the_line = 0;
			for (int ipoint = 0; ipoint < 5 - 1; ipoint++)
			{
				if (areaX[ipoint] != areaX[ipoint + 1] || areaY[ipoint] != areaY[ipoint + 1])
				{
					cross_the_line = 1;
					break;
				}
			}
			double x_updown[4],y_updown[4],z_updown[4];
			for(int i=0;i<2;i++)
			{
				x_updown[i] = x[i];
				y_updown[i] = y[i];
				z_updown[i] = z[i];
			}
			for(int i=2;i<4;i++)
			{
				x_updown[i] = x[i+1];
				y_updown[i] = y[i+1];
				z_updown[i] = z[i+1];
			}
			double a0;
			lsm(z_updown,x_updown,4,a0,slopeX);
			double x3fit = a0+slopeX*z[2];
			lsm(z_updown,y_updown,4,a0,slopeY);
			double y3fit = a0+slopeY*z[2];

			// Calculate delta X and delta Y.
			deltaX = x[2] - x3fit;
			deltaY = y[2] - y3fit;
			deltaTX = tx3 - slopeX;
			deltaTY = ty3 - slopeY;
			x_t = t->X();
			y_t = t->Y();
			plate = pvr->GetPattern(iPID)->Plate();
			tree->Fill();
		}
	}
	return 0;
}
int dedicated_align(TObjArray *tracks,double Xcenter, double Ycenter, double bin_width)
{
	gMinuit = TVirtualFitter::Fitter(0, 300);
	TNtupleD *sta = new TNtupleD("sta", "ShiftTAlign", "iX:iY:shiftX:shiftY:pid");
	int ntrk = tracks->GetEntriesFast();

	for (double iY = Ycenter - XYrange + bin_width / 2; iY <= Ycenter + XYrange; iY += bin_width) // Divide the area into 2*2 mm^2 areas
	{
		for (double iX = Xcenter - XYrange + bin_width / 2; iX <= Xcenter + XYrange; iX += bin_width)
		{
			TObjArray *tracks2 = new TObjArray;

			for (int itrk = 0; itrk < ntrk; itrk++)
			{
				EdbTrackP *t = (EdbTrackP *)tracks->At(itrk);
				// if (t->N()<10|| abs(t->TX() + 0.01) >= 0.01 || abs(t->TY()-0.004) >= 0.01||t->GetSegment(0)->PID()>=10)
				if (t->N() < 10 || abs(t->TX() + 0.01) >= 0.01 || abs(t->TY() - 0.004) >= 0.01)
					continue;
				// if (fabs(t->X() - iX) < bin_width / 2 && fabs(t->Y() - iY) < bin_width / 2)
				if (10 <= count_passed_seg(t, iX, iY, bin_width)) //  check if the track passes the area
					tracks2->Add(t);
			}
			// printf("iX = %.0f, iY = %.0f, ntrk = %d\n", iX, iY, tracks2->GetEntries());
			if (tracks2->GetEntries() == 0)
				continue;
			// calculate the alignment parameters several times.
			for (int j = 0; j < 1; j++)
			{
				calc_align_par(tracks2,iX,iY,bin_width,0,sta); //4th is fixflag
			}
			// Apply alignment parameter.
			for (int itrk = 0; itrk < ntrk; itrk++)
			{
				EdbTrackP *t = (EdbTrackP *)tracks->At(itrk);
				// if (fabs(t->X() - iX) < bin_width / 2 && fabs(t->Y() - iY) < bin_width / 2)
				apply_align(t, iX, iY, bin_width);
			}
			delete tracks2;
		}
	}
	// Ntuple for Shifts of TAlign
	// TFile fout1(Form("ShiftPar/sta_%.0fbinwidth_t1_NoFix_Robust%.1f_NoStraySeg_over10seg.root", bin_width, robustfactor), "recreate");
	// sta->Write();
	// fout1.Close();
	return 0;
}
int main(int argc, char *argv[])
{
	bool alignFlag = true;
	if(argc>5)
	{
		alignFlag = strcmp(argv[5], "--NoAlign") != 0;
	}else{
		printf("Usage: ./calc_dxy linked_tracks.root title Xcenter Ycenter bin_width robustfactor\n");
		printf("Usage: ./calc_dxy linked_tracks.root title Xcenter Ycenter --NoAlign\n");
		return 1;
	}
	if ( argc<7&&alignFlag)
	{
		printf("Usage: ./calc_dxy linked_tracks.root title Xcenter Ycenter bin_width robustfactor\n");
		printf("Usage: ./calc_dxy linked_tracks.root title Xcenter Ycenter --NoAlign\n");
		return 1;
	}

	TString filename_linked_tracks = argv[1];
	TString title = argv[2];
	double Xcenter, Ycenter, bin_width;
	bin_width=20000;
	sscanf(argv[3], "%lf", &Xcenter);
	sscanf(argv[4], "%lf", &Ycenter);
	TString Align_or_NoAlign = "";
	if(!alignFlag) Align_or_NoAlign = "_BeforeAlign";
	
	EdbDataProc *dproc = new EdbDataProc;
	EdbPVRec *pvr = new EdbPVRec;
	dproc->ReadTracksTree(*pvr, filename_linked_tracks, "1");

	nPID = pvr->Npatterns();
	int plMin = pvr->GetPattern(0)->Plate();
	int plMax = pvr->GetPattern(nPID-1)->Plate();

	TObjArray *tracks = pvr->GetTracks();
	int ntrk = tracks->GetEntriesFast();
	
	if (ntrk == 0)
	{
		printf("ntrk==0\n");
		return 0;
	}
	if (alignFlag) //if perform alignment
	{
		sscanf(argv[5], "%lf", &bin_width);
		sscanf(argv[6], "%f", &robustfactor);
		dedicated_align(tracks,Xcenter,Ycenter,bin_width);
	}

	TTree *tree = new TTree("tree", "deltaXY");
	TObjString *info = new TObjString(Form("plMin=%d, plMax=%d, Xcenter=%f, Ycenter=%f", plMin, plMax, Xcenter, Ycenter));
	tree->GetUserInfo()->Add(info);

	calc_dxy(pvr,tree,ntrk,Xcenter,Ycenter,bin_width);
	// Ntuple for deltaXY
	// TFile fout(Form("deltaXY%s_nt_reconnected_%.0fbinwidth_func2_t1.root", module,bin_width), "recreate");
	// TFile fout(Form("deltaXY/nt_aligntfd_NAlign_%.0f_%.0f.root",Xcenter,Ycenter), "recreate");
	// TFile fout(Form("deltaXY/nt_reconnected_%.0fbinwidth_t1_Fix_0to94_Robust%.1f_%.0f_%.0f.root",bin_width,robustfactor,Xcenter,Ycenter), "recreate");
	// TFile fout(Form("deltaXY/nt_aligntfd_reconnected_%.0fbinwidth_NAlign_%.0f_%.0f.root",bin_width,Xcenter,Ycenter), "recreate");
	// TFile fout(Form("deltaXY%s_nt_reconnected_%.0fbinwidth_func2_ttest.root", module,bin_width), "recreate");
	// TFile fout(Form("deltaXY/nt_%.0fbinwidth_t1_NoFix_Robust%.1f_NoStraySeg_over10seg.root",bin_width,robustfactor), "recreate");
	TFile fout("deltaXY/tree_" + title + Align_or_NoAlign + ".root", "recreate");
	tree->Write();
	fout.Close();
	// トラックアライメント後のlinked_tracksを作る
	//  TObjArray *tracks_t = new TObjArray;
	//  for(int itrk=0;itrk<ntrk;itrk++)
	//  {
	//  	EdbTrackP *t = (EdbTrackP *)tracks->At(itrk);
	//  	tracks_t->Add(t);
	//  }
	//  dproc->MakeTracksTree(*tracks_t,0,0,Form("/data/Users/kokui/FASERnu/F222/test/F222_zone3_vertex003_test2/reco43_095000_065000/v13/linked_tracks_AfterAlign_Robust%.1f_NoStraySeg_over10seg_StartAtFirst10.root",robustfactor));

	return 0;
	
}