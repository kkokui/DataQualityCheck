#include "hip/hip_runtime.h"
#include "FnuDivideAlign.h"

#include <stdio.h>
#include <EdbPattern.h>
#include <TFile.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

struct cudaSegment
{
	int flag, pid;
	float x, y, z;
};

struct cudaTrack
{
	float x, y, z, tx, ty, tx_first8, ty_first8;
	int nseg;
	cudaSegment segments[NPIDMAX];
};

float robustFactor = 1.0;
int ncall;
TObjArray *gTracks;

// Data buffer for the GPU process
double *h_params;
double *d_params;
cudaTrack *h_tracks;
cudaTrack *d_tracks;
float *h_chi2;
float *d_chi2;

FnuDivideAlign::FnuDivideAlign()
	: binWidth(2000), rangeXY(8500)
{
}

FnuDivideAlign::~FnuDivideAlign()
{
}

void FnuDivideAlign::SetBinWidth(double bwidth)
{
	binWidth = bwidth;
}

void FnuDivideAlign::SetRobustFactor(float rfactor)
{
	robustFactor = rfactor;
}

double FnuDivideAlign::GetBinWidth()
{
	return binWidth;
}

float FnuDivideAlign::GetRobustFactor()
{
	return robustFactor;
}

__global__ void lsm_kernel(int n, cudaTrack *d_trk, double *d_param)
{
	// Calculation of least square method

	// access thread id
	const unsigned int tid = threadIdx.x;
	const unsigned int tsize = blockDim.x;
	const unsigned int bid = blockIdx.x;
	// x = a0 + a1*z
	int pos = tid + tsize * bid;
	if (pos < n)
	{
		cudaTrack *t = &d_trk[pos];
		int i;
		double A00 = 0, A01 = 0, A02 = 0, A11 = 0, A12 = 0;
		double B00 = 0, B01 = 0, B02 = 0, B11 = 0, B12 = 0;
		for (i = 0; i < NPIDMAX; i++)
		{
			cudaSegment *s = &t->segments[i];
			float x = s->x + d_param[s->pid * 2];
			float y = s->y + d_param[s->pid * 2 + 1];
			float z = s->z;
			if (s->flag)
			{
				A00 += 1.0;
				A01 += z;
				A02 += x;
				A11 += z * z;
				A12 += z * x;
				B00 += 1.0;
				B01 += z;
				B02 += y;
				B11 += z * z;
				B12 += z * y;
			}
		}

		t->x = (A02 * A11 - A01 * A12) / (A00 * A11 - A01 * A01);
		t->tx = (A00 * A12 - A01 * A02) / (A00 * A11 - A01 * A01);

		t->y = (B02 * B11 - B01 * B12) / (B00 * B11 - B01 * B01);
		t->ty = (B00 * B12 - B01 * B02) / (B00 * B11 - B01 * B01);
		t->z = 0;
	}
	__syncthreads();
}

__global__ void calc_chi2_kernel(int n, cudaTrack *d_trk, double *p, float *d_chi2)
{
	// Calculate chi2 of a track

	// access thread id
	const unsigned int tid = threadIdx.x;
	const unsigned int tsize = blockDim.x;
	const unsigned int bid = blockIdx.x;

	int pos = tid + tsize * bid;
	float chi2 = 0;
	if (pos < n)
	{
		float sigmaPos2 = 0.36; // 0.6 * 0.6;
		float sigmaAng2 = 4e-6; // 0.002 * 0.002;

		cudaTrack *t = &d_trk[pos];

		int nseg = 0;
		for (int i = 0; i < NPIDMAX; i++)
		{
			cudaSegment *s = &t->segments[i];
			if (s->flag == 0)
				continue;
			float x = s->x + p[i * 2];
			float y = s->y + p[i * 2 + 1];
			float z = s->z;
			float dx = x - (t->x + t->tx * (z - t->z));
			float dy = y - (t->y + t->ty * (z - t->z));
			chi2 += dx * dx + dy * dy;
			nseg++;
		}
		chi2 /= sigmaPos2 * nseg;
		float txFit = t->tx;
		float tyFit = t->ty;
		float tx = t->tx_first8;
		float ty = t->ty_first8;
		chi2 += ((tx - txFit) * (tx - txFit) + (ty - tyFit) * (ty - tyFit)) / sigmaAng2;
		d_chi2[pos] = chi2;
	}
	__syncthreads();
}

void fitfuncRobust(Int_t &npar, Double_t *grad, Double_t &fval, Double_t *p, Int_t iflag)
{
	// Fit function for TMinuit

	double delta2 = 0.0;

	for (int i = 0; i < NPIDMAX * 2; i++)
	{
		h_params[i] = p[i];
	}

	checkCudaErrors(hipMemcpy(d_params, h_params, sizeof(double) * NPIDMAX * 2, hipMemcpyHostToDevice));

	int ntrk = gTracks->GetEntriesFast();
	int numthread = 512;
	int numblock = (ntrk + numthread - 1) / numthread;
	dim3 threads(numthread, 1, 1);
	dim3 blocks(numblock, 1, 1);

	lsm_kernel<<<blocks, threads>>>(ntrk, d_tracks, d_params);
	hipDeviceSynchronize();
	// check if kernel execution generated and error
	getLastCudaError("lsm Kernel execution failed");

	calc_chi2_kernel<<<blocks, threads>>>(ntrk, d_tracks, d_params, d_chi2);
	hipDeviceSynchronize();
	getLastCudaError("calc chi2 Kernel execution failed");

	// thrust, sort on GPU
	thrust::sort(thrust::device, d_chi2, d_chi2 + ntrk);
	checkCudaErrors(hipMemcpy(h_chi2, d_chi2, sizeof(float) * ntrk, hipMemcpyDeviceToHost));
	int nrobust = ntrk * robustFactor; // For example, if robustFactor = 0.5, only 50% of tracks will be used.
	for (int i = 0; i < nrobust; i++)
	{
		delta2 += h_chi2[i];
	}
	// delta2/=nrobust;

	// regularization
	//  double lambda = 0.1;
	//  for(int i=0;i<nPID*2)
	//  {
	//  	delta2+=lambda*p[i]*p[i]; //L2 regularization
	//  	// delta2+=lambda*abs(p[i]); //L1 regularization
	//  }

	fval = delta2;
	if (ncall % 1000 == 0)
	{
		printf("ncall=%d fval = %lf ", ncall, fval);
		for (int i = 0; i < 10; i++)
		{
			printf("%4.1lf ", p[i]);
		}
		printf("\n");
	}
	ncall++;
}

void FnuDivideAlign::CalcAlignPar(TObjArray *tracks, double iX, double iY, int fixflag)
{
	// Calculate alignment parameters in a divided area

	gTracks = tracks;
	int ntrk = gTracks->GetEntriesFast();
	// Cuda data buffers
	checkCudaErrors(hipHostMalloc((void **)&h_tracks, sizeof(cudaTrack) * ntrk));
	checkCudaErrors(hipHostMalloc((void **)&h_chi2, sizeof(float) * ntrk));
	checkCudaErrors(hipHostMalloc((void **)&h_params, sizeof(double) * NPIDMAX * 2));

	checkCudaErrors(hipMalloc((void **)&d_tracks, sizeof(cudaTrack) * ntrk));
	checkCudaErrors(hipMalloc((void **)&d_chi2, sizeof(float) * ntrk));
	checkCudaErrors(hipMalloc((void **)&d_params, sizeof(double) * NPIDMAX * 2));

	for (int i = 0; i < ntrk; i++)
	{
		// Setup structures for tracks
		EdbTrackP *t = (EdbTrackP *)gTracks->At(i);
		cudaTrack *ct = &h_tracks[i];
		ct->tx_first8 = t->TX();
		ct->ty_first8 = t->TY();
		ct->nseg = t->N();
		for (int ipid = 0; ipid < NPIDMAX; ipid++)
		{
			ct->segments[ipid].flag = 0;
		} // Clear initial values
		for (int iseg = 0; iseg < t->N(); iseg++)
		{
			EdbSegP *s = t->GetSegment(iseg);
			if (fabs(s->X() - iX) < binWidth / 2 && fabs(s->Y() - iY) < binWidth / 2)
			{
				int pid = s->PID();
				ct->segments[pid].flag = 1;
				ct->segments[pid].x = s->X();
				ct->segments[pid].y = s->Y();
				ct->segments[pid].z = s->Z();
			}
		}
	}
	checkCudaErrors(hipMemcpy(d_tracks, h_tracks, sizeof(cudaTrack) * ntrk, hipMemcpyHostToDevice));
	// The default minimizer is Minuit, you can also try Minuit2
	TVirtualFitter::SetDefaultFitter("Minuit");
	// minuit->BuildArrays(30);
	// Int_t SetParameter(Int_t ipar, const char* parname, Double_t value, Double_t verr, Double_t vlow, Double_t vhigh)

	// Set parameters of the initial plate
	int pid = 0; // initial plate
	minuit->SetParameter(2 * pid, Form("dx%d", pid), 0, 0, 0, 0);
	minuit->SetParameter(2 * pid + 1, Form("dy%d", pid), 0, 0, 0, 0);
	// Set parameters of the middle plate
	for (pid = 1; pid < nPID - 1; pid++)
	{
		minuit->SetParameter(2 * pid, Form("dx%d", pid), 0, 0.1, 0, 0);
		minuit->SetParameter(2 * pid + 1, Form("dy%d", pid), 0, 0.1, 0, 0);
	}
	// Set parameters of the last plate
	pid = nPID - 1; // last plate
	if (fixflag == 1)
	{
		minuit->SetParameter(2 * pid, Form("dx%d", pid), 0, 0, 0,0);
		minuit->SetParameter(2 * pid + 1, Form("dy%d", pid), 0, 0, 0,0);
	}
	else
	{
		minuit->SetParameter(2 * pid, Form("dx%d", pid), 0, 0.1, 0, 0);
		minuit->SetParameter(2 * pid + 1, Form("dy%d", pid), 0, 0.1, 0, 0);
	}

	minuit->SetFCN(fitfuncRobust);

	double arglist[200];
	arglist[0] = 0;
	// set print level. arglist[0]==0 is minimum print.
	minuit->ExecuteCommand("SET PRIntout", arglist, 1);

	// minimize
	arglist[0] = 100000; // number of function calls
	arglist[1] = 50;	 // tolerance of estimated verical distance to minimum
	// minuit->SetMaxIterations(10000);
	ncall = 0;
	printf("Aligning iX = %.0f, iY = %.0f, ntrk = %d, robustFactor = %.1f\n", iX, iY, ntrk, robustFactor);
	minuit->ExecuteCommand("MIGRAD2", arglist, 2);

	// get result
	for (int i = 0; i < nPID * 2; ++i)
	{
		p[i] = minuit->GetParameter(i);
		// parErrors[i] = minuit->GetParError(i);
	}

	checkCudaErrors(hipHostFree(h_tracks));
	checkCudaErrors(hipHostFree(h_chi2));
	checkCudaErrors(hipHostFree(h_params));

	checkCudaErrors(hipFree(d_tracks));
	checkCudaErrors(hipFree(d_chi2));
	checkCudaErrors(hipFree(d_params));
}

int FnuDivideAlign::CountPassedSeg(EdbTrackP *t, double iX, double iY)
{
	// Count a number of segments in one track which passed a divided area
	int count = 0;
	for (int iseg = 0; iseg < t->N(); iseg++)
	{
		EdbSegP *s = t->GetSegment(iseg);
		if (fabs(s->X() - iX) < binWidth / 2 && fabs(s->Y() - iY) < binWidth / 2)
			count++;
	}
	return count;
}

void FnuDivideAlign::ApplyAlign(EdbTrackP *t, double iX, double iY)
{
	// Apply alignment to segments which passed a divided area
	for (int iseg = 0; iseg < t->N(); iseg++)
	{
		EdbSegP *s = t->GetSegment(iseg);
		if (fabs(s->X() - iX) < binWidth / 2 && fabs(s->Y() - iY) < binWidth / 2)
		{
			int pid = s->PID();
			s->SetX(s->X() + p[pid * 2]);
			s->SetY(s->Y() + p[pid * 2 + 1]);
		}
	}
}
void FnuDivideAlign::ApplyAlignBicubic(EdbSegP *s,double Xcenter,double Ycenter)
{
	// Apply alignment to segments which passed a divided area

	int pid = s->PID();

	// get shift value calculated by divide align
	// double 

	// get x and y values of the segment
	double segmentPositionX = s->X();
	double segmentPositionY = s->Y();
	// detect nearest middle position of division
	int nearestMiddlePointNumberX = (segmentPositionX - (Xcenter - rangeXY) - binWidth/2) / binWidth;
	int nearestMiddlePointNumberY = (segmentPositionY - (Ycenter - rangeXY) - binWidth/2) / binWidth;
	double nearestMiddlePointPositionX = Xcenter - rangeXY + nearestMiddlePointNumberX*binWidth;
	double nearestMiddlePointPositionY = Ycenter - rangeXY + nearestMiddlePointNumberY*binWidth;
	// detect 16 reference values
	double referencePositionX[4][4];
	double referencePositionY[4][4];

	double referenceShiftX[4][4];
	double referenceShiftY[4][4];
	// if(segmentPositionX<nearestBinPositionX)
	// {

	// 	referenceShiftX[0][0] = 
	// }
	// calculate alignment parameter
	// apply alignment

}
int FnuDivideAlign::Align(TObjArray *tracks, double Xcenter, double Ycenter, int nPatterns)
{
	// Divide the area, Calculate alignment parameters and apply alignment
	nPID = nPatterns;
	minuit = TVirtualFitter::Fitter(0, nPID * 2);
	alignPar = new TTree("alignPar", "alignPar");
	alignPar->Branch("iX", &iXBranchValue);
	alignPar->Branch("iY", &iYBranchValue);
	alignPar->Branch("shiftX", &shiftXBranchValue);
	alignPar->Branch("shiftY", &shiftYBranchValue);
	alignPar->Branch("pid", &pidBranchValue);

	// int nDivisionX = rangeXY*2/binWidth +1;
	// int nDivisionY = rangeXY*2/binWidth +1;
	// double ***shiftXEachDivision = new double**[nPatterns];
	// double ***shiftYEachDivision = new double**[nPatterns];
	// for(int pid=0;pid<nPatterns;pid++)
	// {
	// 	shiftXEachDivision[pid] = new double*[nDivisionX];
	// 	shiftYEachDivision[pid] = new double*[nDivisionX];
	// 	for(int iDivisionX=0;iDivisionX<nDivisionX;iDivisionX++)
	// 	{
	// 		shiftXEachDivision[pid][iDivisionX] = new double[nDivisionY];
	// 		shiftYEachDivision[pid][iDivisionX] = new double[nDivisionY];
	// 	}
	// }

	int ntrk = tracks->GetEntriesFast();

	double angleXSum = 0;
	double angleYSum = 0;
	for (int itrk = 0; itrk < ntrk; itrk++)
	{
		EdbTrackP *t = (EdbTrackP*)tracks->At(itrk);
		angleXSum += t->TX();
		angleYSum += t->TY();
	}
	double angleXMean = angleXSum/ntrk;
	double angleYMean = angleYSum/ntrk;

	// Divide the area into binWidth*binWidth um^2 areas
	for (iYBranchValue = Ycenter - rangeXY + binWidth / 2; iYBranchValue <= Ycenter + rangeXY; iYBranchValue += binWidth)
	// max of iY is not correct sometimes...
	{
		for (iXBranchValue = Xcenter - rangeXY + binWidth / 2; iXBranchValue <= Xcenter + rangeXY; iXBranchValue += binWidth)
		{
			TObjArray *tracks2 = new TObjArray;

			for (int itrk = 0; itrk < ntrk; itrk++)
			{
				EdbTrackP *t = (EdbTrackP *)tracks->At(itrk);
				if (t->N() < 10 || abs(t->TX() -angleXMean) >= 0.01 || abs(t->TY() - angleYMean) >= 0.01)
				// if (t->N() < 10 )
				{
					continue;
				}
				if (10 <= CountPassedSeg(t, iXBranchValue, iYBranchValue)) //  check if the track passes the area
				{
					tracks2->Add(t);
				}
			}
			if (tracks2->GetEntries() < 20)
			{
				continue;
			}

			// calculate the alignment parameters several times.
			for (int j = 0; j < 1; j++)
			{
				CalcAlignPar(tracks2, iXBranchValue, iYBranchValue, 0);
			}

			int iXID = (iXBranchValue - (Xcenter - rangeXY + binWidth / 2))/binWidth;
			int iYID = (iYBranchValue - (Ycenter - rangeXY + binWidth / 2))/binWidth;
			for (pidBranchValue = 0; pidBranchValue < nPID; pidBranchValue++)
			{
				shiftXBranchValue = p[pidBranchValue * 2];
				shiftYBranchValue = p[pidBranchValue * 2 + 1];
				alignPar->Fill();
			}
			for (int itrk = 0; itrk < ntrk; itrk++)
			{
				EdbTrackP *t = (EdbTrackP *)tracks->At(itrk);
				ApplyAlign(t, iXBranchValue, iYBranchValue);
			}
			delete tracks2;
		}
	}
	return 0;
}

void FnuDivideAlign::WriteAlignPar(TString filename)
{
	// Write TTree for Shifts of alignment
	TFile fout1(filename, "recreate");
	alignPar->Write();
	fout1.Close();
}
