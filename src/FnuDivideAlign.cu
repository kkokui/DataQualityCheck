#include "hip/hip_runtime.h"
#include "FnuDivideAlign.h"

#include <EdbDataSet.h>
#include <TObjArray.h>
#include <TVirtualFitter.h>
#include <TMath.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>


float robustFactor;
int ncall = 0;
TObjArray *gTracks;

double *h_params;
double *d_params;
cudaTrack* d_tracks;
float* d_chi2;
float* h_chi2;

FnuDivideAlign::FnuDivideAlign()
{
	binWidth = 2000;
	robustFactor = 1.0;
	XYrange = 8500;
}

FnuDivideAlign::~FnuDivideAlign()
{}

void FnuDivideAlign::SetBinWidth(double bwidth)
{
	binWidth = bwidth;
}

void FnuDivideAlign::SetRobustFactor(float rfactor)
{
	robustFactor = rfactor;
}

double FnuDivideAlign::GetBinWidth()
{
	return binWidth;
}

float FnuDivideAlign::GetRobustFactor()
{
	return robustFactor;
}

// 最小二乗法の計算 


__global__ void lsm_kernel(int n, cudaTrack* d_trk, double *d_param)
{
	// access thread id
	const unsigned int tid = threadIdx.x;
	const unsigned int tsize = blockDim.x;
	const unsigned int bid = blockIdx.x;
	// y = a0 + a1*x
	// x = a0 + a1*z
	int pos = tid + tsize*bid;
	if (pos < n) {
		cudaTrack *t = &d_trk[pos];
        int i;
        double A00=0 ,A01=0, A02=0, A11=0, A12=0;
        double B00=0 ,B01=0, B02=0, B11=0, B12=0;
        for (i=0;i<NPIDMAX;i++) {
			cudaSegment *s = &t->segments[i];
	 		float x = s->x + d_param[s->pid*2];
	 		float y = s->y + d_param[s->pid*2+1];
	 		float z = s->z;
			if(s->flag){
                A00+=1.0;
                A01+=z;
                A02+=x;
                A11+=z*z;
                A12+=z*x;
                B00+=1.0;
                B01+=z;
                B02+=y;
                B11+=z*z;
                B12+=z*y;
            }
        }
 
        t->x = (A02*A11-A01*A12) / (A00*A11-A01*A01);
        t->tx = (A00*A12-A01*A02) / (A00*A11-A01*A01);

        t->y = (B02*B11-B01*B12) / (B00*B11-B01*B01);
        t->ty = (B00*B12-B01*B02) / (B00*B11-B01*B01);
        t->z = 0;
	}
	__syncthreads();
}

__global__ void calc_chi2_kernel(int n, cudaTrack* d_trk, double *p, float *d_chi2)
{
	// access thread id
	const unsigned int tid = threadIdx.x;
	const unsigned int tsize = blockDim.x;
	const unsigned int bid = blockIdx.x;
	
	int pos = tid + tsize*bid;
	float chi2 = 0;
	if (pos < n) {
		float sigmaPos2 = 0.36;//0.6 * 0.6;
		float sigmaAng2 = 4e-6;//0.002 * 0.002;
		
		cudaTrack *t = &d_trk[pos];
		
		int nseg=0;
		for(int i=0; i<NPIDMAX; i++)
		{
			cudaSegment *s = &t->segments[i];
			if(s->flag==0) continue;
			float x = s->x + p[i*2];
			float y = s->y + p[i*2+1];
			float z = s->z;
			float dx = x - (t->x+t->tx*(z-t->z));
			float dy = y - (t->y+t->ty*(z-t->z));
			chi2 += dx * dx + dy * dy;
			nseg++;
		}
		chi2 /= sigmaPos2*nseg;
		float txFit = t->tx;
		float tyFit = t->ty;
		float tx = t->tx_first8;
		float ty = t->ty_first8;
		chi2 += ((tx - txFit) * (tx - txFit) + (ty - tyFit) * (ty - tyFit)) / sigmaAng2;
		d_chi2[pos] = chi2;
		
	}
	__syncthreads();
}


void fitfuncRobust(Int_t &npar, Double_t *grad, Double_t &fval, Double_t *p, Int_t iflag)
{
	
	double delta2 = 0.0;
	
	for(int i=0; i<NPIDMAX*2; i++){h_params[i]=p[i];}
	
	checkCudaErrors( hipMemcpy( d_params, h_params, sizeof(double)*NPIDMAX*2, hipMemcpyHostToDevice) );
	
	int ntrk = gTracks->GetEntriesFast();
	int numthread = 512;
	int numblock = (ntrk + numthread -1)/numthread;
	dim3 threads(numthread, 1, 1);
	dim3 blocks(numblock, 1, 1);
	
	lsm_kernel <<< blocks, threads >>> (ntrk, d_tracks, d_params);
	hipDeviceSynchronize();
	// check if kernel execution generated and error
	getLastCudaError("lsm Kernel execution failed");
	
	/*if(ncall==0){
		checkCudaErrors( hipMemcpy( h_tracks, d_tracks, sizeof(cudaTrack)*ntrk, hipMemcpyDeviceToHost) );
		printf("h_track %f %f %f %f %f\n", h_tracks[0].x, h_tracks[0].y, h_tracks[0].z, h_tracks[0].tx,  h_tracks[0].ty);
	}*/
	
	calc_chi2_kernel<<< blocks, threads >>> (ntrk, d_tracks, d_params, d_chi2);
	hipDeviceSynchronize();
	getLastCudaError("calc chi2 Kernel execution failed");
	
	// thrust, sort on GPU
	thrust::sort(thrust::device, d_chi2, d_chi2+ntrk);
	checkCudaErrors( hipMemcpy( h_chi2, d_chi2, sizeof(float)*ntrk, hipMemcpyDeviceToHost) );
	// float robustFactor = 0.5; // only % of segments will be used.
	int nrobust = ntrk*robustFactor;
	for (int i = 0; i < nrobust; i++) 
	{
		delta2 += h_chi2[i];
	}
	
	//regularization
	// double lambda = 0.1;
	// for(int i=0;i<nPID*2)
	// {
	// 	delta2+=lambda*p[i]*p[i]; //L2 regularization
	// 	// delta2+=lambda*abs(p[i]); //L1 regularization
	// }
	fval = delta2;
	if(ncall%1000==0) {
		printf("ncall=%d fval = %lf ", ncall, fval);
		for(int i=0; i<10; i++){printf("%4.1lf ", p[i]);}
		printf("\n");
	}
	ncall++;
}

void FnuDivideAlign::calc_align_par(TObjArray *tracks,double iX, double iY, int fixflag)
{
	gTracks = tracks;
	int ntrk = gTracks->GetEntriesFast();
	// Cuda data buffers
	checkCudaErrors( hipHostMalloc( (void**) &h_tracks, sizeof(cudaTrack)*ntrk) );
	checkCudaErrors( hipHostMalloc( (void**) &h_chi2, sizeof(float)*ntrk) );
	checkCudaErrors( hipHostMalloc( (void**) &h_params, sizeof(double)*NPIDMAX*2) );
	
	checkCudaErrors( hipMalloc( (void**) &d_tracks, sizeof(cudaTrack)*ntrk) );
	checkCudaErrors( hipMalloc( (void**) &d_chi2, sizeof(float)*ntrk) );
	checkCudaErrors( hipMalloc( (void**) &d_params, sizeof(double)*NPIDMAX*2) );
	
	for (int i = 0; i < ntrk; i++)
	{
		EdbTrackP *t = (EdbTrackP *)gTracks->At(i);
		cudaTrack *ct = &h_tracks[i];
		ct->tx_first8 = t->TX();
		ct->ty_first8 = t->TY();
		ct->nseg = t->N();
		for(int ipid=0; ipid<NPIDMAX; ipid++){ ct->segments[ipid].flag=0;} //初期値クリア
		for (int iseg = 0; iseg < t->N(); iseg++)
		{
			EdbSegP *s = t->GetSegment(iseg);
			// if (1)
			if (fabs(s->X() - iX) < binWidth / 2 && fabs(s->Y() - iY) < binWidth / 2)
			{
				int pid = s->PID();
				ct->segments[pid].flag=1;
				ct->segments[pid].x=s->X();
				ct->segments[pid].y=s->Y();
				ct->segments[pid].z=s->Z();
			}
		}
	}
	checkCudaErrors( hipMemcpy( d_tracks, h_tracks, sizeof(cudaTrack)*ntrk, hipMemcpyHostToDevice) );
	// The default minimizer is Minuit, you can also try Minuit2
	TVirtualFitter::SetDefaultFitter("Minuit");
	// gMinuit->BuildArrays(30);
	// Int_t SetParameter(Int_t ipar, const char* parname, Double_t value, Double_t verr, Double_t vlow, Double_t vhigh)

	int pid=0; //最初のプレート
	gMinuit->SetParameter(2*pid,Form("dx%d",pid),0,0,0,0);
	gMinuit->SetParameter(2*pid+1,Form("dy%d",pid),0,0,0,0);
	for(pid=1;pid<nPID-1;pid++)
	{
		gMinuit->SetParameter(2*pid,Form("dx%d",pid),0,0.1,-30,30);
		gMinuit->SetParameter(2*pid+1,Form("dy%d",pid),0,0.1,-30,30);
	}
	pid=nPID-1; // 最後のプレート
	if(fixflag==1)
	{
		gMinuit->SetParameter(2*pid,Form("dx%d",pid),0,0,0,0);
		gMinuit->SetParameter(2*pid+1,Form("dy%d",pid),0,0,0,0);
	}else{
		gMinuit->SetParameter(2*pid,Form("dx%d",pid),0,0.1,-30,30);
		gMinuit->SetParameter(2*pid+1,Form("dy%d",pid),0,0.1,-30,30);
	}
	
	
	gMinuit->SetFCN(fitfuncRobust);
	
	
	double arglist[200];

	arglist[0] = 0;
	// set print level. arglist[0]==0 is minimum print.
	gMinuit->ExecuteCommand("SET PRIntout",arglist,1);

	// minimize
	arglist[0] = 200000; // number of function calls
	arglist[1] = 0.001; // tolerance
	gMinuit->SetMaxIterations( 10000 );
	ncall =0;
	printf("Aligning iX = %.0f, iY = %.0f, ntrk = %d, robustFactor = %.1f\n",iX,iY,ntrk,robustFactor);
	gMinuit->ExecuteCommand("MIGRAD2", arglist, 2);
	/*
	gMinuit->SetFCN(fitfuncRobust3);
	gMinuit->ExecuteCommand("MIGRAD",arglist,2);
	*/
	
	/*
	double p[3];
	double parErrors[3];
	*/
	// get result
	for (int i = 0; i < nPID*2; ++i)
	{
		p[i] = gMinuit->GetParameter(i);

		// parErrors[i] = minuit->GetParError(i);
	}
	
	
	checkCudaErrors( hipHostFree( h_tracks) );
	checkCudaErrors( hipHostFree( h_chi2) );
	checkCudaErrors( hipHostFree( h_params) );

	checkCudaErrors( hipFree( d_tracks) );
	checkCudaErrors( hipFree( d_chi2) );
	checkCudaErrors( hipFree( d_params) );
	

}


int FnuDivideAlign::count_passed_seg(EdbTrackP *t, double iX, double iY)
{
	int count = 0;
	for (int iseg = 0; iseg < t->N(); iseg++)
	{
		EdbSegP *s = t->GetSegment(iseg);
		if (fabs(s->X() - iX) < binWidth / 2 && fabs(s->Y() - iY) < binWidth / 2)
			count++;
	}
	return count;
}
void FnuDivideAlign::apply_align(EdbTrackP *t, double iX, double iY)
{
	for (int iseg = 0; iseg < t->N(); iseg++)
	{
		EdbSegP *s = t->GetSegment(iseg);
		// if (1)
		if (fabs(s->X() - iX) < binWidth / 2 && fabs(s->Y() - iY) < binWidth / 2)
		{
			int pid = s->PID();
			s->SetX(s->X() + p[pid * 2]);
			s->SetY(s->Y() + p[pid * 2 + 1]);
		}
	}
}

int FnuDivideAlign::dedicated_align(EdbPVRec *pvr,double Xcenter, double Ycenter)
{
	TObjArray *tracks = pvr->GetTracks();
	nPID = pvr->Npatterns();
	int plMin = pvr->GetPattern(0)->Plate();
	int plMax = pvr->GetPattern(nPID-1)->Plate();
	gMinuit = TVirtualFitter::Fitter(0, 300);
	alignPar = new TTree("alignPar","alignPar");
	double iX, iY, shiftX, shiftY;
	int pid;
	alignPar->Branch("iX",&iX);
	alignPar->Branch("iY",&iY);
	alignPar->Branch("shiftX",&shiftX);
	alignPar->Branch("shiftY",&shiftY);
	alignPar->Branch("pid",&pid);
	int ntrk = tracks->GetEntriesFast();

	for (iY = Ycenter - XYrange + binWidth / 2; iY <= Ycenter + XYrange; iY += binWidth) // Divide the area into 2*2 mm^2 areas
	{
		for (iX = Xcenter - XYrange + binWidth / 2; iX <= Xcenter + XYrange; iX += binWidth)
		{
			TObjArray *tracks2 = new TObjArray;

			for (int itrk = 0; itrk < ntrk; itrk++)
			{
				EdbTrackP *t = (EdbTrackP *)tracks->At(itrk);
				// if (t->N()<10|| abs(t->TX() + 0.01) >= 0.01 || abs(t->TY()-0.004) >= 0.01||t->GetSegment(0)->PID()>=10)
				if (t->N() < 10 || abs(t->TX() + 0.01) >= 0.01 || abs(t->TY() - 0.004) >= 0.01)
					continue;
				// if (fabs(t->X() - iX) < binWidth / 2 && fabs(t->Y() - iY) < binWidth / 2)
				if (10 <= count_passed_seg(t, iX, iY)) //  check if the track passes the area
					tracks2->Add(t);
			}
			// printf("iX = %.0f, iY = %.0f, ntrk = %d\n", iX, iY, tracks2->GetEntries());
			if (tracks2->GetEntries() == 0)
				continue;
			// calculate the alignment parameters several times.
			for (int j = 0; j < 1; j++)
			{
				calc_align_par(tracks2,iX,iY,0); //4th is fixflag
			}
			
			for(pid=0;pid<nPID;pid++)
			{
				shiftX = p[pid*2];
				shiftY = p[pid*2+1];
				alignPar->Fill();
			}
			// Apply alignment parameter.
			for (int itrk = 0; itrk < ntrk; itrk++)
			{
				EdbTrackP *t = (EdbTrackP *)tracks->At(itrk);
				// if (fabs(t->X() - iX) < binWidth / 2 && fabs(t->Y() - iY) < binWidth / 2)
				apply_align(t, iX, iY);
			}
			delete tracks2;
		}
	}
	return 0;
}

void FnuDivideAlign::WriteAlignPar(TString filename)
{
	//Write TTree for Shifts of alignment
	TFile fout1(filename, "recreate");
	alignPar->Write();
	fout1.Close();
}
